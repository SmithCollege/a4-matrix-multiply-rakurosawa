
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>

// compile as: nvcc cublasMM.cu -o cublasMM -lcublas

int main(){
    int size = 1000;

    float *x, *y, *z;
    hipMallocManaged(&x, sizeof(float) * size * size);
    hipMallocManaged(&y, sizeof(float) * size * size);
    hipMallocManaged(&z, sizeof(float) * size * size);

    // initialize values for x and y arrays
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            x[i * size + j] = 1; // x[i][j]
            y[i * size + j] = 1;
        }
    }
    
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, size, size, size, &alpha, x, size, y, size, &beta, z, size);

    hipblasDestroy(handle);

    hipDeviceSynchronize();

        for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            if (z[i * size + j] != size) {
                printf("Error at z[%d][%d]: %f\n", i, j, z[i * size + j]);
            }
        }
    }

    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}
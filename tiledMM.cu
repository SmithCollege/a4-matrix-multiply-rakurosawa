
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define TILE_WIDTH 16 // 16 * 16 = 256

__global__ void MatMul(float* d_M, float* d_N, float* d_P, int Width){
    __shared__ float subTileM[TILE_WIDTH][TILE_WIDTH];
    __shared__ float subTileN[TILE_WIDTH][TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = blockIdx.y*TILE_WIDTH+ty;
    int Col = blockIdx.x*TILE_WIDTH+tx;
    float Pvalue = 0;

    for (int m = 0; m < Width/TILE_WIDTH; m++){
        subTileM[ty][tx] = d_M[Row * Width + m * TILE_WIDTH + tx];
        subTileN[ty][tx] = d_N[(m * TILE_WIDTH + ty) * Width + Col];
        __syncthreads();
        for (int k = 0; k < TILE_WIDTH; k++){
            float value1, value2;
            value1 = subTileM[ty][k];
            value2 = subTileN[k][tx];
            Pvalue += value1 * value2;
            d_P[Row * Width + Col] = + Pvalue;
        }
        __syncthreads();
    }
}

int main(){
    int size = 128; // test 128, 256, 512

    // allocate memory for the arrays
    float *x, *y, *z;
    hipMallocManaged(&x, sizeof(float) * size * size);
    hipMallocManaged(&y, sizeof(float) * size * size);
    hipMallocManaged(&z, sizeof(float) * size * size);

    // initialize values for x and y arrays
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            x[i * size + j] = 1; // x[i][j]
            y[i * size + j] = 1;
        }
    }

    // calculate a "2D" thread and block call
    dim3 dimGrid(ceil((1.0*size)/TILE_WIDTH), ceil((1.0*size)/TILE_WIDTH), 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    MatMul<<<dimGrid, dimBlock>>>(x, y, z, size);
    hipDeviceSynchronize();

    // // print the output for small size testing
    // for (int i = 0; i < size; i++){
    //     for (int j = 0; j < size; j++){
    //         printf("[%f] ", z[i * size + j]);
    //     }
    //     printf("\n");
    // }

    // check for errors (presumes array is only 1's)
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            if (z[i * size + j] != size) {
                printf("Error at z[%d][%d]: %f\n", i, j, z[i * size + j]);
            }
        }
    }

    // free memory
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}